#include "hip/hip_runtime.h"
#include "image.h"

#define PIXELS_PER_BLOCK 750
#define THREAD_PER_BLOCK 25

#define min(a, b) ((a) < (b) ? (a) : (b))

__global__ void median_filter(BYTE* img, int filter_size, int height, int width, int channels, BYTE* res)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    BYTE* medians_arrays = (BYTE*) malloc(sizeof(BYTE) * filter_size * filter_size * channels);

    filter_size /= 2;

    for (int i = x * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK);
         i < min(height,
                 x * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK) + PIXELS_PER_BLOCK / THREAD_PER_BLOCK);
         i++)
    {
        for (int j = y * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK);
             j < min(width, y * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK)
                         + PIXELS_PER_BLOCK / THREAD_PER_BLOCK);
             j++)
        {
            for (int c = 0; c < channels; c++)
            {
                for (int v = -filter_size; v <= filter_size; v++)
                {
                    for (int h = -filter_size; h <= filter_size; h++)
                    {
                        if (i + v >= 0 && j + h >= 0 && i + v <= height - 1 && j + h <= width - 1)
                            medians_arrays[c * ((filter_size * 2 + 1) * (filter_size * 2 + 1))
                                + (filter_size + v) * (filter_size * 2 + 1) + (h + filter_size)]
                                = img[(i + v) * width * channels + (j + h) * channels + c];
                        else
                            medians_arrays[c * ((filter_size * 2 + 1) * (filter_size * 2 + 1))
                                + (filter_size + v) * (filter_size * 2 + 1) + (h + filter_size)]
                                = 0;
                    }
                }
		
                int x, z, m, aux;
                BYTE * array = medians_arrays + c * ((filter_size * 2 + 1) * (filter_size * 2 + 1));
                int size = (filter_size * 2 + 1) * (filter_size * 2 + 1);

                for (x = 0; x < (size - 1); x++)
                {
                    m = x;
                    for (z = (x + 1); z < size; z++)
                    {
                        if (array[z] < array[m])
                        {
                            m = z;
                        }
                    }
                    if (x != m)
                    {
                        aux = array[x];
                        array[x] = array[m];
                        array[m] = aux;
                    }
                }
	
                res[i * width * channels + j * channels + c]
                    = medians_arrays[c * ((filter_size * 2 + 1) * (filter_size * 2 + 1))
                        + filter_size * (filter_size * 2 + 1) + filter_size];
            }
        }
    }

    free(medians_arrays);
}


int main(int argc, char** argv)
{
    if (argc != 4)
    {
        printf("usage : %s <input_image_path> <filter_size> <output_image_path\n", argv[0]);
	exit(1);
    }

    int filter_size = atoi(argv[2]);
    const char * input = argv[1];
    const char * output = argv[3];
    int width, height, channels;
    BYTE* h_img = stbi_load(input, &width, &height, &channels, 0);
    
    BYTE* d_img;
    BYTE* d_res_img;
    if (h_img == NULL)
    {
        printf("Error in loading the image\n");
        exit(1);
    }

    printf("THREAD_PER_BLOCK : %d \nPIXELSE_PER_BLOCK : %d\n", THREAD_PER_BLOCK, PIXELS_PER_BLOCK);
    printf("Loaded image with \nheight : %d \nwidth : %d \nchannels : %d\n", height, width, channels);
    
    hipMalloc(&d_img, sizeof(BYTE) * width * height * channels);
    hipMalloc(&d_res_img, sizeof(BYTE) * width * height * channels);

    hipMemcpy(d_img, h_img, sizeof(BYTE) * width * height * channels, hipMemcpyHostToDevice);


    int blck_x = (height + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK;
    int blck_y = (width + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK;
    printf("blck x    %d    blck y    %d  \n", blck_x, blck_y);
    median_filter<<<dim3(blck_x, blck_y), dim3(THREAD_PER_BLOCK, THREAD_PER_BLOCK)>>>(d_img, filter_size, height, width, channels, d_res_img);

    hipMemcpy(h_img, d_res_img, sizeof(BYTE) * height * width * channels, hipMemcpyDeviceToHost);

    if (!stbi_write_jpg(output, width, height, channels, h_img, 100))
    {
        printf("Error in writing the image\n");
        exit(1);
    }

    free(h_img);
    hipFree(d_img);
    hipFree(d_res_img);

    return 0;
}
