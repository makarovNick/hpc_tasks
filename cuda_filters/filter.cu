#include "hip/hip_runtime.h"
#include "image.h"

#define PIXELS_PER_BLOCK 512
#define THREAD_PER_BLOCK 4

#define min(a, b) ((a) < (b) ? (a) : (b))

__global__ void filter(BYTE* img, int filter_size, double* filter, int height, int width, int channels, BYTE* res)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    double s = 0.0;
    //if (x ==0 && y == 0)
    {
        for (int i = 0; i < filter_size; i++)
        {
            for (int j = 0; j < filter_size; j++)
            {
                s += filter[i * filter_size + j];
            }
        }

        for (int i = 0; i < filter_size; i++)
        {
            for (int j = 0; j < filter_size; j++)
            {
                filter[i * filter_size + j] /= s;
            }
        }
        filter_size /= 2;
    }

    //__syncthreads();
    //printf("%d , %lf \n", filter_size, s);    
    for (int i = x * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK); i < min(height, x * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK) + PIXELS_PER_BLOCK / THREAD_PER_BLOCK); i++)
    {
        for (int j = y * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK); j < min(width, y * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK) + PIXELS_PER_BLOCK / THREAD_PER_BLOCK); j++)
        {
            for (int c = 0; c < channels; c++)
            {
            	res[i * width * channels + j * channels + c] = 0;
                for (int v = -filter_size; v <= filter_size; v++)
                {
                    for (int h = -filter_size; h <= filter_size; h++)
                    {
                        if (i + v >= 0 && j + h >= 0 && i + v <= height - 1 && j + h <= width - 1)
                        {
                            res[i * width * channels + j * channels + c] += filter[(v + filter_size) * (filter_size * 2 + 1) + h + filter_size] * img[(i + v) * width * channels + (j + h) * channels + c];
                        }
                    }
                }

            }
        }
    }
}


int main(int argc, char** argv) 
{
    
    if (argc < 3)
    {
	printf("usage : %s <input_image_path> <output_image_path>", argv[0]);
	exit(1);
    }
    int width, height, channels;
    BYTE* h_img = stbi_load(argv[1], &width, &height, &channels, 0);
    BYTE* d_img;
    BYTE* d_res_img;
    if(h_img == NULL) 
    {
        printf("Error in loading the image\n");
        exit(1);
    }

    hipMalloc(&d_img, sizeof(BYTE) * width * height * channels);
    hipMalloc(&d_res_img, sizeof(BYTE) * width * height * channels);

    hipMemcpy(d_img, h_img, sizeof(BYTE) * width * height *channels, hipMemcpyHostToDevice);

    int filter_size = 3;
    printf("Size of filter : \n");
    scanf("%d", &filter_size);
    double* h_filter = (double*) malloc(sizeof(double) * filter_size * filter_size);
    if (filter_size & 1 == 0)
    {
	printf("filter size must be odd\n");
        exit(1);
    }
    printf("Filter : \n");
    for (int i = 0; i < filter_size; ++i)
    {
        for (int j = 0; j < filter_size; ++j)
	{
	    printf("filter[i][j] : \n");
	    scanf("%d", &h_filter[i * filter_size + j]);
	}
    }
    double* d_filter;

    // h_filter[0] = 1;     h_filter[1] = 1;   h_filter[2] = 1;
    // h_filter[3] = 1;     h_filter[4] = 1;   h_filter[5] = 1;
    // h_filter[6] = 1;     h_filter[7] = 1;   h_filter[8] = 1;

    hipMalloc(&d_filter, sizeof(double) * filter_size * filter_size);
    hipMemcpy(d_filter, h_filter, sizeof(double) * filter_size * filter_size, hipMemcpyHostToDevice);

    int blck_x = (height + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK;
    int blck_y = (width + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK;
    filter<<<dim3(blck_x, blck_y), dim3(THREAD_PER_BLOCK, THREAD_PER_BLOCK)>>>(d_img, filter_size, d_filter, height, width, channels, d_res_img);

    hipMemcpy(h_img, d_res_img, sizeof(BYTE) * height * width * channels, hipMemcpyDeviceToHost);

    if(!stbi_write_jpg(argv[2], width, height, channels, h_img, 100))
    {
        printf("Error in writing the image\n");
        exit(1);
    }

    free(h_filter);
    hipFree(d_filter);
    free(h_img);
    hipFree(d_img);
    hipFree(d_res_img);
   

    return 0;
}
