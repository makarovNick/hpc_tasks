#include "hip/hip_runtime.h"
#include "image.h"

#define PIXELS_PER_BLOCK 512
#define THREAD_PER_BLOCK 4

#define min(a, b) ((a) < (b) ? (a) : (b))

__global__ void histogram(BYTE* img, int height, int width, int channels, int * histogram)
{

    int x = threadIdx.x + blockIdx.x * blockDim.x; 
    int y = threadIdx.y + blockIdx.y * blockDim.y; 
    
    for (int i = x * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK); i < min(height, x * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK) + PIXELS_PER_BLOCK / THREAD_PER_BLOCK); i++)
    {
        for (int j = y * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK); j < min(width, y * (PIXELS_PER_BLOCK / THREAD_PER_BLOCK) + PIXELS_PER_BLOCK / THREAD_PER_BLOCK); j++)
        {
            if (channels < 3)
            {
		
                atomicAdd(histogram + img[i * width * channels + j * channels + 0], 1);
            }
            else
            {
                atomicAdd(histogram + (int)(img[i * width * channels + j * channels + 0] * 0.299
                                          + img[i * width * channels + j * channels + 1] * 0.587
                                          + img[i * width * channels + j * channels + 2] * 0.114), 1);
            } 
        }
    }
}

void save_histogram(const char* name, int* hist, int height)
{    
    BYTE* img = (BYTE*) malloc(sizeof(BYTE) * height * 256);
    int max = 0;
    for (int i = 0; i < 256; i++)
    {
        max = (max > hist[i]) ? max : hist[i];
    }

    double coef = 1.0 * height / max;
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < 256; j++)
        {
            if ((int)(hist[j] * coef) <= i)
            {
                img[(height - 1 - i) * 256 + j + 0] = 0;
            }
            else
            {
                img[(height - 1 - i) * 256 + j + 0] = 255;
            }
        }
    }

    stbi_write_jpg(name, 256, height, 1, img, 100);

    free(img);
}

int main(int argc, char** argv) 
{
    int width, height, channels;
    BYTE* h_image =  stbi_load(argv[1], &width, &height, &channels, 0);
    printf("H = %d, W = %d, C = %d\n", height, width, channels);
    BYTE* d_image;
    int* d_out;
    int* h_hist =(int*) malloc(sizeof(int) * 256);

    hipMalloc(&d_image, height * width * height * sizeof(BYTE));
    hipMalloc(&d_out, 256 * sizeof(int));

    hipMemcpy(d_image, h_image, sizeof(BYTE) * channels * width * height, hipMemcpyHostToDevice);

    int blck_x = (height + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK;
    int blck_y = (width + PIXELS_PER_BLOCK - 1) / PIXELS_PER_BLOCK;
    
    memset(h_hist, '\0', sizeof(int) * 256);
    hipMemcpy(d_out, h_hist, sizeof(int) * 256, hipMemcpyHostToDevice);
    histogram<<<dim3(blck_x, blck_y), dim3(THREAD_PER_BLOCK, THREAD_PER_BLOCK)>>>(d_image, height, width, channels, d_out);
    hipMemcpy(h_hist, d_out, sizeof(int) * 256 , hipMemcpyDeviceToHost);
    for (int i = 0; i < 256; ++i)
    {
	printf("%d ", h_hist[i]);
    }
 
    save_histogram("hist.jpg", h_hist, 100);

    free(h_image);
    free(h_hist);
    hipFree(d_out);
    hipFree(d_image);

    return 0;
}
